#include "hip/hip_runtime.h"
#include"stdio.h"
#include <string.h>
#include <stdlib.h>
#include <time.h>
#include"math.h"
#include <ctype.h>
//int *tmp;
__host__ __device__ int distD(int i,int j,float *x,float*y)
{
	int dx=x[i]-x[j];
	int dy=y[i]-y[j]; 
	return(sqrtf( (dx*dx) + (dy*dy) ));
}

/*kernel function to generate the feasible solution and determine the best improvement among them */
__global__ void tsp(float *pox,float *poy,int cost,unsigned long long *dst_tid,int cit)
{

	int i,j;
	register int change=0;
	long sol=(cit)*(cit-1)/2;
	int id=threadIdx.x+blockIdx.x*blockDim.x;
	if(id<sol)
	{
		
		i=cit-2-floorf(((int)__dsqrt_rn(8*(sol-id-1)+1)-1)/2);
		j=id-i*(cit-1)+(i*(i+1)/2)+1;
		if(i<j && i<cit && j<cit && i!=j-1)
		{
			change=distD(i,j,pox,poy)+distD((i+1)%cit,(j+1)%cit,pox,poy)-distD(i,(i+1)%cit,pox,poy)-distD(j,(j+1)%cit,pox,poy);
			cost+=change;	
			if(change < 0)
				 atomicMin(dst_tid, ((unsigned long long)cost << 32) | id);
		}
		if(i<0 || j <0 || i>=cit ||j>=cit)
			printf("\nD id=%d i=%d j=%d cost=%d change:%d\n",id,i,j,cost,change);
	}
	
}

/*Arrange x,y coordinates in route order*/
void twoOpt(int x,int y,float *pox,float *poy)
{
	float *tmp_x,*tmp_y;
	int i,j;
	tmp_x=(float*)malloc(sizeof(float)*(y-x));	
	tmp_y=(float*)malloc(sizeof(float)*(y-x));	
	for(j=0,i=y;i>x;i--,j++)
	{
		tmp_x[j]=pox[i];
		tmp_y[j]=poy[i];
	}
	for(j=0,i=x+1;i<=y;i++,j++)
	{
		pox[i]=tmp_x[j];
		poy[i]=tmp_y[j];
	}
	free(tmp_x);
	free(tmp_y);

}
/*check the whether initiated route is valid or not*/
void routeChecker(long N)
{
	int *b,i,j,flag=0;
	
	b=(int*)calloc(N,sizeof(int));	
	FILE* file = fopen ("input", "r");
	if(file==NULL)
		printf("null");
	j = 0;
	for(; fscanf(file, "%d", &j) && !feof(file);)
	{
		b[j]++;
	}
	fclose(file);
	for(i=0;i<N;i++)
	{
		if(b[i]>1 || b[i] ==0)
		{
			flag=1;printf("breaking at %d",i);break;
		}
	}
	if(flag==1)
	{
		printf("\nroute is not valid");

	}
	else
	{
		printf("\nroute is valid");
	}

}


void setCoord(int *r,float *posx,float *posy,float *px,float *py,int cities)
{

	for(int i=0;i<cities;i++)
	{
	px[i]=posx[r[i]];

	py[i]=posy[r[i]];
	
	}
}

static void CudaTest(char *msg)
{
	hipError_t e;
	hipDeviceSynchronize();
	if(hipSuccess != (e = hipGetLastError()))
	{
		fprintf(stderr, "%s: %d\n", msg, e);
		fprintf(stderr, "%s\n", hipGetErrorString(e));
		exit(-1);
	}
}
 
int main(int argc, char *argv[])
{
	int ch, cnt, in1, cities;
	float in2, in3;
	FILE *f;
	float *posx, *posy;
	float *px, *py;
	char str[256];  
	float *d_posx, *d_posy;
	int dst,d,tid;
        unsigned long long *d_dst_tid;
	int x,y;
	int blk,thrd;
	clock_t start,end;
	long sol;
	int *r,i,j;
	f = fopen(argv[1], "r");
	if (f == NULL) {fprintf(stderr, "could not open file \n");  exit(-1);}

	ch = getc(f);  while ((ch != EOF) && (ch != '\n')) ch = getc(f);
	ch = getc(f);  while ((ch != EOF) && (ch != '\n')) ch = getc(f);
	ch = getc(f);  while ((ch != EOF) && (ch != '\n')) ch = getc(f);

	ch = getc(f);  while ((ch != EOF) && (ch != ':')) ch = getc(f);
	fscanf(f, "%s\n", str);
	cities = atoi(str);
	if (cities <= 2) {fprintf(stderr, "only %d cities\n", cities);  exit(-1);}

	sol=cities*(cities-1)/2;
	posx = (float *)malloc(sizeof(float) * cities);  if (posx == NULL) {fprintf(stderr, "cannot allocate posx\n");  exit(-1);}
	posy = (float *)malloc(sizeof(float) * cities);  if (posy == NULL) {fprintf(stderr, "cannot allocate posy\n");  exit(-1);}
	px = (float *)malloc(sizeof(float) * cities);  if (posx == NULL) {fprintf(stderr, "cannot allocate posx\n");  exit(-1);}
	py = (float *)malloc(sizeof(float) * cities);  if (posy == NULL) {fprintf(stderr, "cannot allocate posy\n");  exit(-1);}
	r = (int *)malloc(sizeof(int) * cities);  if (posy == NULL) {fprintf(stderr, "cannot allocate posy\n");  exit(-1);}
	
	ch = getc(f);  while ((ch != EOF) && (ch != '\n')) ch = getc(f);
	fscanf(f, "%s\n", str);
	if (strcmp(str, "NODE_COORD_SECTION") != 0) {fprintf(stderr, "wrong file format\n");  exit(-1);}

	cnt = 0;

	while (fscanf(f, "%d %f %f\n", &in1, &in2, &in3)) 
	{
		posx[cnt] = in2;
		posy[cnt] = in3;
		cnt++;
		if (cnt > cities) {fprintf(stderr, "input too long\n");  exit(-1);}
		if (cnt != in1) {fprintf(stderr, "input line mismatch: expected %d instead of %d\n", cnt, in1);  exit(-1);}
	}

	if (cnt != cities) {fprintf(stderr, "read %d instead of %d cities\n", cnt, cities);  exit(-1);}
	fscanf(f, "%s", str);
	if (strcmp(str, "EOF") != 0) {fprintf(stderr, "didn't see 'EOF' at end of file\n");  exit(-1);}
    	fflush(f);
	fclose(f);
	/*threads and blocks setting up*/
	if(sol<=50000)
	{
	blk=(sol-1)/512+1;
	thrd=512;
	}
	else
	{
	blk=(sol-1)/1024+1;
	thrd=1024;
	}

//----------------------------------------------------
/*initial route generation and its cost calculation*/
	r[0]=0;
	int k=1;i=0;float min;int minj,mini,count=1,flag=0;dst=0;
	int *v=(int*)calloc(cities,sizeof(int));
	v[0]=1;
	while(count!=cities)
	{
		flag=0;
		for(j=1;j<cities;j++)
		{
			if(i!=j && !v[j])
			{
				min=distD(i,j,posx,posy);
				minj=j;
				break;	
			}
		}

		for(j=minj+1;j<cities;j++)
		{
			
				 if( !v[j])
				{
					if(min>distD(i,j,posx,posy))
					{
						min=distD(i,j,posx,posy);
						mini=j;
						flag=1;				
					}
				}
		}
		if(flag==0)
			i=minj;
		else
			i=mini;
		dst+=min;
		r[k++]=i;v[i]=1;
		count++;
	}
	free(v);
	dst+=distD(r[0],r[cities-1],posx,posy);
//-------------------------------------------------
	FILE *fp2=fopen("input","w");
	for(int i=0;i<cities;i++)
	{
		fprintf(fp2,"%d\n",r[i]);
	}
	fclose(fp2);
	routeChecker(cities);

	setCoord(r,posx,posy,px,py,cities);
	count=0;
	start = clock();
	hipEvent_t strt, stp;
	hipEventCreate(&strt);
	hipEventCreate(&stp);
 	unsigned long long dst_tid = (((long)dst+1) << 32) -1;
        unsigned long long dtid;

	printf("\ninitial cost : %d\n",dst);
	if(hipSuccess!=hipMalloc((void**)&d_posx,sizeof(float)*cities))printf("\nAllocating memory for x on GPU");
	if(hipSuccess!=hipMalloc((void**)&d_posy,sizeof(float)*cities))printf("\nAllocating memory for y on GPU");
	if(hipSuccess!=hipMalloc((void**)&d_dst_tid,sizeof(unsigned long long)))printf("\nAllocating memory for dst_tid on GPU");
    	if(hipSuccess!=hipMemcpy(d_dst_tid,&dst_tid,sizeof(unsigned long long),hipMemcpyHostToDevice))printf("\ntransfer on GPU");
	if(hipSuccess!=hipMemcpy(d_posx,px,sizeof(float)*cities,hipMemcpyHostToDevice))printf("\ntransfer on GPU");
	if(hipSuccess!=hipMemcpy(d_posy,py,sizeof(float)*cities,hipMemcpyHostToDevice))printf("\ntransfer on GPU");
	hipEventRecord(strt,0);	
	tsp<<<blk,thrd>>>(d_posx,d_posy,dst,d_dst_tid,cities);
	CudaTest("kernel launch failed");
	hipEventRecord(stp,0);		hipEventSynchronize(stp);
	if(hipSuccess!=hipMemcpy(&dtid,d_dst_tid,sizeof(unsigned long long),hipMemcpyDeviceToHost))
	printf("\nCan't transfer minimal cost back to CPU");

	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, strt, stp);
  	d = dtid >> 32;
	printf("\nfirst cost found %d",d);	
	while( d < dst )
	{
		dst=d;
		tid = dtid & ((1ull<<32)-1); 
		x=cities-2-floor((sqrt(8*(sol-tid-1)+1)-1)/2);
		y=tid-x*(cities-1)+(x*(x+1)/2)+1;
		twoOpt(x,y,px,py);
		if(hipSuccess!=hipMemcpy(d_posx,px,sizeof(float)*cities,hipMemcpyHostToDevice))printf("\ntransfer on GPU");
		if(hipSuccess!=hipMemcpy(d_posy,py,sizeof(float)*cities,hipMemcpyHostToDevice))printf("\ntransfer on GPU");
		unsigned long long dst_tid = (((long)dst+1) << 32) -1;
		if(hipSuccess!=hipMemcpy(d_dst_tid,&dst_tid,sizeof(unsigned long long),hipMemcpyHostToDevice))
		printf("\ntransfer on GPU");

		tsp<<<blk,thrd>>>(d_posx,d_posy,dst,d_dst_tid,cities);
		CudaTest("kernel launch failed");
		if(hipSuccess!=hipMemcpy(&dtid,d_dst_tid,sizeof(unsigned long long),hipMemcpyDeviceToHost))
		printf("\nCan't transfer minimal cost back to CPU");
	  	d = dtid >> 32;
		count++;
	}
	printf("\nMinimal Distance : %d\n",d);

	printf("\nnumber of time climbed %d\n",count);
	end = clock();
	double t=((double) (end - start)) / CLOCKS_PER_SEC;
	printf("\ntime : %f\n",t);
	
	hipFree(d_posy);
	hipFree(d_posx);
	hipFree(d_dst_tid);
	free(posx);
	free(posy);
	free(px);
	free(py);
	free(r);
	return 0;
}
