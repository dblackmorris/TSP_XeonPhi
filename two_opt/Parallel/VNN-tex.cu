#include "hip/hip_runtime.h"
#include"stdio.h"
#include <string.h>
#include <stdlib.h>
#include <time.h>
#include"math.h"
#include <ctype.h>
//int *tmp;
texture<float, 1, hipReadModeElementType> t_posx;
texture<float, 1, hipReadModeElementType> t_posy;

int distNN(int i,int j,float *x,float*y)
{
	unsigned int dx=x[i]-x[j];
	unsigned int dy=y[i]-y[j]; 
	return(sqrtf( (dx*dx) + (dy*dy) ));
}

__device__ int distD(int i,int j)
{
int dx=tex1D(t_posx,i)-tex1D(t_posx,j);
int dy=tex1D(t_posy,i)-tex1D(t_posy,j); 
return(sqrtf( (dx*dx) + (dy*dy) ));
}


__global__ void tsp(int cost,unsigned long long *dst_tid,int cit)
{

	int i,j;
	register int change=0;
	long sol=(cit)*(cit-1)/2;
	int id=threadIdx.x+blockIdx.x*blockDim.x;
	if(id<sol)
	{
		
		i=cit-2-floorf(((int)__dsqrt_rn(8*(sol-id-1)+1)-1)/2);
		j=id-i*(cit-1)+(i*(i+1)/2)+1;
		if(i<j && i<cit && j<cit && i!=j-1)
		{
		change=distD(i,j)+distD((i+1)%cit,(j+1)%cit)-distD(i,(i+1)%cit)-distD(j,(j+1)%cit);
		cost+=change;	
		if(change < 0)
			 atomicMin(dst_tid, ((unsigned long long)cost << 32) | id);
		}
		if(i<0 || j <0 || i>=cit ||j>=cit)
			printf("\nD id=%d i=%d j=%d cost=%d change:%d\n",id,i,j,cost,change);
	}
	
}

int distH(float *px,float *py,int cit)
{
	int dx,dy,cost=0;
	int i;
	for(i=0;i<(cit-1);i++)
	{
		dx=px[i]-px[i+1];
		dy=py[i]-py[i+1]; 
		cost+=sqrtf( (dx*dx) + (dy*dy) );

	}
	dx=px[i]-px[0];
	dy=py[i]-py[0]; 
	cost+=sqrtf( (dx*dx) + (dy*dy) );

	return cost;

}

void twoOpt(int x,int y,float *pox,float *poy)
{
		float *tmp_x,*tmp_y;
		int i,j;
		tmp_x=(float*)malloc(sizeof(float)*(y-x));	
		tmp_y=(float*)malloc(sizeof(float)*(y-x));	
		for(j=0,i=y;i>x;i--,j++)
		{
			tmp_x[j]=pox[i];
			tmp_y[j]=poy[i];
		}
		for(j=0,i=x+1;i<=y;i++,j++)
		{
			pox[i]=tmp_x[j];
			poy[i]=tmp_y[j];
		}
		free(tmp_x);
		free(tmp_y);

}


void setCoord(int *r,float *posx,float *posy,float *px,float *py,int cities)
{

	for(int i=0;i<cities;i++)
	{
	px[i]=posx[r[i]];

	py[i]=posy[r[i]];
	
	}
}

static void CudaTest(char *msg)
{
	hipError_t e;
	hipDeviceSynchronize();
	if (hipSuccess != (e = hipGetLastError()))
	{
		fprintf(stderr, "%s: %d\n", msg, e);
		fprintf(stderr, "%s\n", hipGetErrorString(e));
		exit(-1);
	}
}
 
int main(int argc, char *argv[])
{
	int ch, cnt, in1, cities;
	float in2, in3;
	FILE *f;
	float *posx, *posy;
	float *px, *py;
	char str[256];  
	int dst,d,tid;
        unsigned long long *d_dst_tid;
	int x,y;
	int blk,thrd;
	clock_t start,end;
	long sol;
	int *r,i,j;
	f = fopen(argv[1], "r");
	if (f == NULL) {fprintf(stderr, "could not open file \n");  exit(-1);}

	ch = getc(f);  while ((ch != EOF) && (ch != '\n')) ch = getc(f);
	ch = getc(f);  while ((ch != EOF) && (ch != '\n')) ch = getc(f);
	ch = getc(f);  while ((ch != EOF) && (ch != '\n')) ch = getc(f);

	ch = getc(f);  while ((ch != EOF) && (ch != ':')) ch = getc(f);
	fscanf(f, "%s\n", str);
	cities = atoi(str);
	if (cities <= 2) {fprintf(stderr, "only %d cities\n", cities);  exit(-1);}

	sol=cities*(cities-1)/2;
	posx = (float *)malloc(sizeof(float) * cities);  if (posx == NULL) {fprintf(stderr, "cannot allocate posx\n");  exit(-1);}
	posy = (float *)malloc(sizeof(float) * cities);  if (posy == NULL) {fprintf(stderr, "cannot allocate posy\n");  exit(-1);}
	px = (float *)malloc(sizeof(float) * cities);  if (posx == NULL) {fprintf(stderr, "cannot allocate posx\n");  exit(-1);}
	py = (float *)malloc(sizeof(float) * cities);  if (posy == NULL) {fprintf(stderr, "cannot allocate posy\n");  exit(-1);}
	r = (int *)malloc(sizeof(int) * cities);  if (posy == NULL) {fprintf(stderr, "cannot allocate posy\n");  exit(-1);}
	
	ch = getc(f);  while ((ch != EOF) && (ch != '\n')) ch = getc(f);
	fscanf(f, "%s\n", str);
	if (strcmp(str, "NODE_COORD_SECTION") != 0) {fprintf(stderr, "wrong file format\n");  exit(-1);}

	cnt = 0;

	while (fscanf(f, "%d %f %f\n", &in1, &in2, &in3)) 
	{
		posx[cnt] = in2;
		posy[cnt] = in3;
		cnt++;
		if (cnt > cities) {fprintf(stderr, "input too long\n");  exit(-1);}
		if (cnt != in1) {fprintf(stderr, "input line mismatch: expected %d instead of %d\n", cnt, in1);  exit(-1);}
	}

	if (cnt != cities) {fprintf(stderr, "read %d instead of %d cities\n", cnt, cities);  exit(-1);}
	fscanf(f, "%s", str);
	if (strcmp(str, "EOF") != 0) {fprintf(stderr, "didn't see 'EOF' at end of file\n");  exit(-1);}
    	fflush(f);
	fclose(f);
	if(sol<=50000)
	{
		blk=(sol-1)/512+1;
		thrd=512;
	}
	else
	{
		blk=(sol-1)/1024+1;
		thrd=1024;
	}

//----------------------------------------------------
	r[0]=0;
	int k=1;i=0;float min;int minj,mini,count=1,flag=0;dst=0;
	int *v=(int*)calloc(cities,sizeof(int));
	v[0]=1;
	while(count!=cities)
	{
		flag=0;
		for(j=1;j<cities;j++)
		{
			if(i!=j && !v[j])
			{
				min=distNN(i,j,posx,posy);
				minj=j;
				break;	
			}
		}

		for(j=minj+1;j<cities;j++)
		{
			
				 if( !v[j])
				{
					if(min>distNN(i,j,posx,posy))
					{
						min=distNN(i,j,posx,posy);
						mini=j;
						flag=1;				
					}
				}
		}
		if(flag==0)
			i=minj;
		else
			i=mini;
		dst+=min;
		r[k++]=i;v[i]=1;
		count++;
	}
	free(v);
	dst+=distNN(0,r[cities-1],posx,posy);

//----------------------------------------------------
	setCoord(r,posx,posy,px,py,cities);
	count=0;
	start = clock();
	hipEvent_t strt, stp;
	hipEventCreate(&strt);
	hipEventCreate(&stp);
 	unsigned long long dst_tid = (((long)dst+1) << 32) -1;
        unsigned long long dtid;

	hipArray* d_posx = NULL; 
	hipMallocArray(&d_posx, &t_posx.channelDesc, cities, 1); 
	hipMemcpyToArray(d_posx, 0, 0, px, cities * sizeof(float), hipMemcpyHostToDevice); 
	hipBindTextureToArray(t_posx, d_posx); 
	t_posx.normalized = false; 
	t_posx.addressMode[0] = hipAddressModeClamp;

	hipArray* d_posy = NULL; 
	hipMallocArray(&d_posy, &t_posy.channelDesc, cities, 1); 
	hipMemcpyToArray(d_posy, 0, 0, py, cities * sizeof(float), hipMemcpyHostToDevice); 
	hipBindTextureToArray(t_posy, d_posy); 
	t_posy.normalized = false; 
	t_posy.addressMode[0] = hipAddressModeClamp;

	printf("\ninitial cost : %d\n",dst);
	if(hipSuccess!=hipMalloc((void**)&d_dst_tid,sizeof(unsigned long long)))printf("\nAllocating memory for dst_tid on GPU");
    	if(hipSuccess!=hipMemcpy(d_dst_tid,&dst_tid,sizeof(unsigned long long),hipMemcpyHostToDevice))printf("\ntransfer on GPU");
	hipEventRecord(strt,0);	
	tsp<<<blk,thrd>>>(dst,d_dst_tid,cities);
	CudaTest("kernel launch failed");
	hipEventRecord(stp,0);		hipEventSynchronize(stp);
	if(hipSuccess!=hipMemcpy(&dtid,d_dst_tid,sizeof(unsigned long long),hipMemcpyDeviceToHost))
	printf("\nCan't transfer minimal cost back to CPU");

	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, strt, stp);
  	d = dtid >> 32;
	printf("\nfirst cost found %d ",d);	
	while( d < dst )
	{
		dst=d;
		tid = dtid & ((1ull<<32)-1); 
		x=cities-2-floor((sqrt(8*(sol-tid-1)+1)-1)/2);
		y=tid-x*(cities-1)+(x*(x+1)/2)+1;
		twoOpt(x,y,px,py);

		unsigned long long dst_tid = (((long)dst+1) << 32) -1;
		if(hipSuccess!=hipMemcpy(d_dst_tid,&dst_tid,sizeof(unsigned long long),hipMemcpyHostToDevice))
		printf("\ntransfer on GPU");
		
		hipMemcpyToArray(d_posx, 0, 0, px, cities * sizeof(float), hipMemcpyHostToDevice); 
		hipBindTextureToArray(t_posx, d_posx); 

		hipMemcpyToArray(d_posy, 0, 0, py, cities * sizeof(float), hipMemcpyHostToDevice); 
		hipBindTextureToArray(t_posy, d_posy); 

		tsp<<<blk,thrd>>>(dst,d_dst_tid,cities);
		CudaTest("kernel launch failed");
		if(hipSuccess!=hipMemcpy(&dtid,d_dst_tid,sizeof(unsigned long long),hipMemcpyDeviceToHost))
		printf("\nCan't transfer minimal cost back to CPU");
	  	d = dtid >> 32;
		count++;
	}
	printf("\nMinimal Distance : %d\n",d);

	printf("\nnumber of time climbed %d\n",count);
	end = clock();
	double t=((double) (end - start)) / CLOCKS_PER_SEC;
	printf("\ntime : %f\n",t);
	long long climb=1LL*count * sol;
	printf("\ngmoves/sec :%f\n",climb * 0.000000001 / t);
	hipFree(d_posy);
	hipFree(d_posx);
	hipFree(d_dst_tid);
	free(posx);
	free(posy);
	free(px);
	free(py);
	free(r);
	return 0;
}
