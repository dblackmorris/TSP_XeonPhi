
#include <hip/hip_runtime.h>
#include"stdio.h"
#include <string.h>
#include <stdlib.h>
#include <time.h>
#include"math.h"
#include <ctype.h>
//int *tmp;
__host__  __device__ int distD(int x,int y,int N,float*dt)
{
	int id;
	id=x*N+y;
	return(dt[id]);
}

/*kernel function to generate the feasible solution and determine the best improvement among them */
__global__ void tsp(int *rt,int cost,unsigned long long *dst_tid,int cit,float *dt)
    {

    	int i,j;
    	register int change=0;
    	int sol=(cit)*(cit-1)/2;
    	int id=threadIdx.x+blockIdx.x*blockDim.x;
	
    	if(id<sol)
    	{
    		i=cit-2-floorf(((int)__dsqrt_rn(8*(sol-id-1)+1)-1)/2);
    		j=id-i*(cit-1)+(i*(i+1)/2)+1;
		if(i<j && i<cit && j<cit && i!=j-1)
		{
		
			change=distD(rt[i],rt[j],cit,dt)+distD(rt[(i+1)%cit],rt[(j+1)%cit],cit,dt)-distD(rt[i],rt[(i+1)%cit],cit,dt)-distD(rt[j],rt[(j+1)%cit],cit,dt);
			cost+=change;
			if (change < 0)
				atomicMin(dst_tid, ((unsigned long long)cost << 32) | id);
    		}
	}
    	
    }
/*calculate cost of specified route*/
int distH(float *px,float *py,int cit)
{
	float dx,dy,cost=0;
	int i;
	for(i=0;i<(cit-1);i++)
	{
		dx=px[i]-px[i+1];
		dy=py[i]-py[i+1]; 
		cost+=sqrtf( (dx*dx) + (dy*dy) );

	}
	dx=px[i]-px[0];
	dy=py[i]-py[0]; 
	cost+=sqrtf( (dx*dx) + (dy*dy) );

	return cost;

}

 int* twoOpt(int x,int y,int *route,int city)
    {
    	int *tmp;
    	tmp=(int *)malloc(sizeof(int )*city);
    	int i,j;

    	for (i = 0; i <=x; ++i)
    	{
    		tmp[i] = route[i];
    	}

    	for (i = x+1, j = y; i <= y; ++i, --j)
    	{
    		tmp[i] = route[j];
    	}


    	for (i = y+1; i < city; ++i)
    	{
    		tmp[i] = route[i];
    	}


    	return tmp;

    }



void setCoord(int *r,float *posx,float *posy,float *px,float *py,int cities)
{

	for(int i=0;i<cities;i++)
	{
		px[i]=posx[r[i]];

		py[i]=posy[r[i]];
	
	}
}
 
int main(int argc, char *argv[])
{
	int ch, cnt, in1, cities;
	float in2, in3;
	FILE *f;
	float *posx, *posy;

	char str[256];  
	int dst,d,tid;
        unsigned long long *d_dst_tid;
	int x,y;
	int blk,thrd;
	clock_t start,end;
	long sol;
	int *r,i,j;
	f = fopen(argv[1], "r");
	if (f == NULL) {fprintf(stderr, "could not open file \n");  exit(-1);}

	ch = getc(f);  while ((ch != EOF) && (ch != '\n')) ch = getc(f);
	ch = getc(f);  while ((ch != EOF) && (ch != '\n')) ch = getc(f);
	ch = getc(f);  while ((ch != EOF) && (ch != '\n')) ch = getc(f);

	ch = getc(f);  while ((ch != EOF) && (ch != ':')) ch = getc(f);
	fscanf(f, "%s\n", str);
	cities = atoi(str);
	if (cities <= 2) {fprintf(stderr, "only %d cities\n", cities);  exit(-1);}

	sol=cities*(cities-1)/2;
	posx = (float *)malloc(sizeof(float) * cities);  if (posx == NULL) {fprintf(stderr, "cannot allocate posx\n");  exit(-1);}
	posy = (float *)malloc(sizeof(float) * cities);  if (posy == NULL) {fprintf(stderr, "cannot allocate posy\n");  exit(-1);}
	r = (int *)malloc(sizeof(int) * cities);  if (posy == NULL) {fprintf(stderr, "cannot allocate posy\n");  exit(-1);}
	ch = getc(f);  while ((ch != EOF) && (ch != '\n')) ch = getc(f);
	fscanf(f, "%s\n", str);
	if (strcmp(str, "NODE_COORD_SECTION") != 0) {fprintf(stderr, "wrong file format\n");  exit(-1);}

	cnt = 0;

	while (fscanf(f, "%d %f %f\n", &in1, &in2, &in3)) 
	{
		posx[cnt] = in2;
		posy[cnt] = in3;
		cnt++;
		if (cnt > cities) {fprintf(stderr, "input too long\n");  exit(-1);}
		if (cnt != in1) {fprintf(stderr, "input line mismatch: expected %d instead of %d\n", cnt, in1);  exit(-1);}
	}

	if (cnt != cities) {fprintf(stderr, "read %d instead of %d cities\n", cnt, cities);  exit(-1);}
	fscanf(f, "%s", str);
	if (strcmp(str, "EOF") != 0) {fprintf(stderr, "didn't see 'EOF' at end of file\n");  exit(-1);}
    	fflush(f);
	fclose(f);
	/*thread and block setting up*/
	if(sol<=50000)
	{
		blk=(sol-1)/512+1;
		thrd=512;
	}
	else
	{
		blk=(sol-1)/1024+1;
		thrd=1024;
	}
	/*generate distance matrix*/
	float *dist_mat;
	dist_mat = (float *)malloc(sizeof(float) * (cities*cities));
	for (int i = 0; i < cities; ++i)
	{
		for (int j = 0; j < cities; ++j)
		{
		dist_mat[i*cities+j] = sqrtf(pow(posx[i] - posx[j], 2)
		             +powf(posy[i] - posy[j], 2));
		//k++;		
		}
	}
//----------------------------------------------------
/*initial route generation and its cost calculation*/
	r[0]=0;
	int k=1;i=0;float min;int minj,mini,count=1,flag=0;dst=0;
	int *v=(int*)calloc(cities,sizeof(int));
	v[0]=1;
	while(count!=cities)
	{
		flag=0;
		for(j=1;j<cities;j++)
		{
			if(i!=j && !v[j])
			{
				int id;
				if(i>j)
				{id=j*cities+i;}
				else{id=i*cities+j;}	
				min=dist_mat[id];
				minj=j;
				break;	
			}
		}

		for(j=minj+1;j<cities;j++)
		{
			
				 if( !v[j])
				{
					int id;
				if(i>j)
				{id=j*cities+i;}
				else{id=i*cities+j;}	
					if(min>dist_mat[id])
					{
						min=dist_mat[id];
						mini=j;
						flag=1;				
					}
				}
		}
		if(flag==0)
			i=minj;
		else
			i=mini;
		dst+=min;
		r[k++]=i;v[i]=1;
		count++;
	}
	free(v);
	dst+=dist_mat[r[cities-1]];
//-----------------------------------------------------------
	start = clock();
 	unsigned long long dst_tid = (((long)dst+1) << 32) -1;
        unsigned long long dtid;
	int *d_r;
    	float *d_mt;
	printf("\ninitial cost : %d\n",dst);
	/*Allocating memory on GPU */
	if(hipSuccess!=hipMalloc((void**)&d_dst_tid,sizeof(unsigned long long)))printf("\nAllocating memory for dst_tid on GPU");
	if(hipSuccess!=hipMalloc((void**)&d_mt,sizeof(float)*(cities*cities)))printf("\nAllocating memory for thread id on GPU");
    	if(hipSuccess!=hipMalloc((void**)&d_r,sizeof(int)*cities))printf("\nAllocating memory for thread id on GPU");
	/*Data transfer on GPU */
    	if(hipSuccess!=hipMemcpy(d_dst_tid,&dst_tid,sizeof(unsigned long long),hipMemcpyHostToDevice))printf("\ntransfer on GPU");
	if(hipSuccess!=hipMemcpy(d_mt,dist_mat,sizeof(float)*(cities*cities),hipMemcpyHostToDevice))printf("\ntransfer on GPU 1");
    	if(hipSuccess!=hipMemcpy(d_r,r,sizeof(int)*cities,hipMemcpyHostToDevice))printf("\ntransfer on GPU 1");

	tsp<<<blk,thrd>>>(d_r,dst,d_dst_tid,cities,d_mt);

    	if(hipSuccess!=hipMemcpy(&dtid,d_dst_tid,sizeof(unsigned long long),hipMemcpyDeviceToHost))
	printf("\nCan't transfer minimal cost back to CPU");
        d = dtid >> 32;
    	count =0;
	/*This loop contineous until no improvement possible*/
    	while( d < dst )
    	{
    		dst=d;
	        tid = dtid & ((1ull<<32)-1); 
	    	x=cities-2-floor((sqrt(8*(sol-tid-1)+1)-1)/2);
	    	y=tid-x*(cities-1)+(x*(x+1)/2)+1;
	    	r=twoOpt(x,y,r,cities);

                unsigned long long dst_tid = (((long)dst+1) << 32) -1;
    		if(hipSuccess!=hipMemcpy(d_r,r,sizeof(int)*cities,hipMemcpyHostToDevice))printf("\ntransfer on GPU 1");
    	        if(hipSuccess!=hipMemcpy(d_dst_tid,&dst_tid,sizeof(unsigned long long),hipMemcpyHostToDevice))
		printf("\ntransfer on GPU");

    		tsp<<<blk,thrd>>>(d_r,dst,d_dst_tid,cities,d_mt);

    	        if(hipSuccess!=hipMemcpy(&dtid,d_dst_tid,sizeof(unsigned long long),hipMemcpyDeviceToHost))
                printf("\nCan't transfer minimal cost back to CPU");
    		d = dtid >> 32;
                count++;
    	}
printf("\nMinimal Distance : %d\n",d);

printf("\nnumber of time climbed %d\n",count);
end = clock();
double t=((double) (end - start)) / CLOCKS_PER_SEC;
printf("\ntime : %f\n",t);

hipFree(d_dst_tid);
free(posx);
free(posy);
free(r);
return 0;
}
